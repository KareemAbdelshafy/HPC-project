#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

__host__ void init_vects(int vect_len,float *h_vect1,float *h_vect2);
__global__ void vec_add(int vect_len, float *d_vect1, float *d_vect2, float *d_sum);

int main(int argc,char **argv)
{
hipEvent_t start=0;
hipEvent_t stop=0;
float time=0;
hipEventCreate(&start);
hipEventCreate(&stop);

int vect_len=pow(2,18);
float h_vect1[vect_len], h_vect2[vect_len], h_sum[vect_len];
float *d_vect1, *d_vect2, *d_sum;
// initialization
init_vects(vect_len, h_vect1, h_vect2);

// tranfer vectors to global memory
hipMalloc((void **)&d_vect1 , vect_len*sizeof(float) );
hipMalloc((void **)&d_vect2 , vect_len*sizeof(float) );
hipMalloc((void **)&d_sum , vect_len*sizeof(float) );

hipMemcpy (d_vect1 , h_vect1 , vect_len*sizeof(float) , hipMemcpyHostToDevice);
hipMemcpy (d_vect2 , h_vect2 , vect_len*sizeof(float) , hipMemcpyHostToDevice);

// determine block and grid size.

dim3 DimGrid((vect_len/1024),1 ,1);
dim3 DimBlock(1024,1,1);


hipEventRecord(start,0);
vec_add<<<DimGrid,DimBlock>>>(vect_len, d_vect1 ,d_vect2 ,d_sum);
hipEventRecord(stop,0);
hipEventSynchronize(stop);

hipMemcpy(h_sum , d_sum , vect_len*sizeof(float) , hipMemcpyDeviceToHost);
//Free the Device array
hipFree (d_vect1);
hipFree (d_vect2);
hipFree (d_sum);
hipEventElapsedTime(&time,start,stop);
printf("time of the Kernel %f \n",time );

printf("v1=%f ,, v2 =%f ,, sum=%f \n",h_vect1[0],h_vect2[0],h_sum[0]);

return 0;
}

__global__ void vec_add(int vect_len, float *d_vect1, float *d_vect2, float *d_sum){

int tid = blockIdx.x * blockDim.x + threadIdx.x ;

//if(tid<vect_len)
d_sum[tid]= d_vect1[tid] + d_vect2[tid];

}


__host__ void init_vects(int vect_len,float *h_vect1,float *h_vect2){
srand(time(NULL));
 for (int i=0; i<vect_len; i++){
    h_vect1[i] = rand();
    h_vect2[i] = rand();
  }
}
