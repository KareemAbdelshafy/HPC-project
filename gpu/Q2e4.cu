#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define BLOCK_SIZE 1024 // You can change this
//#define NUM_OF_ELEMS 1e6 // You can change this


__global__  void total(float * input, float * output, int len) 
{
    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    // Traverse reduction tree
   for (unsigned int stride = len/2; stride > 0; stride /= 2)
    {
        if ((tid < stride) && (input[tid] > input[tid + stride]) ){
           input[tid] = input[tid + stride];}

    __syncthreads();
    }   

    // Write the computed sum of the block to the output vector at correct index
    if (tid == 0)
    {
        output[0] = input[0];
    }
}

int main(int argc, char ** argv) 
{
    float * hostInput; // The input 1D vector
    float * hostOutput; // The output vector
    float * deviceInput;
    float * deviceOutput;
    int NUM_OF_ELEMS=pow(2,18);

hipEvent_t start=0;
hipEvent_t stop=0;
float timef=0;
hipEventCreate(&start);
hipEventCreate(&stop);

    int numInputElements = NUM_OF_ELEMS; // number of elements in the input list
    //int numOutputElements; // number of elements in the output list
    hostInput = (float *) malloc(sizeof(float) * numInputElements);

    srand(time(NULL));
    for (int i=0; i < NUM_OF_ELEMS; i++)
    {
        hostInput[i] = rand();  
    }

    printf("host %f and %f and %f \n", hostInput[10] , hostInput[20] , hostInput[30]);

    hostOutput = (float*) malloc(sizeof(float));

    //@@ Allocate GPU memory here
    hipMalloc((void **)&deviceInput, numInputElements * sizeof(float));
    hipMalloc((void **)&deviceOutput, sizeof(float));

    // Copy memory to the GPU here
    hipMemcpy(deviceInput, hostInput, numInputElements * sizeof(float), hipMemcpyHostToDevice);

    // Initialize the grid and block dimensions here
  

    dim3 DimGrid(ceil(NUM_OF_ELEMS/1024), 1, 1);
    dim3 DimBlock(BLOCK_SIZE, 1, 1);

    // Launch the GPU Kernel here
    hipEventRecord(start,0);
    total<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, numInputElements);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    // Copy the GPU memory back to the CPU here
    hipMemcpy(hostOutput, deviceOutput, sizeof(float), hipMemcpyDeviceToHost);

     printf("Reduced Sum from GPU = %f \n", hostOutput[0]);  
     hipEventElapsedTime(&timef,start,stop);
     printf("time of the Kernel %f \n",timef );  

    // Free the GPU memory here
    hipFree(deviceInput);
    hipFree(deviceOutput); 
    free(hostInput);
    free(hostOutput);

    return 0;
}

